#include "hip/hip_runtime.h"
#include <iostream>
#include <time.h>
#include <vector>
#include <sys/time.h>
#include <cstring>

#include "tracing/vec3.h"
#include "tracing/ray.h"
#include "tracing/camera.h"
#include "tracing/triangle.h"
#include "tracing/stlobject.h"
#include "tracing/stlparser.h"
#include "acceleration/kdtree.h"
#include "tracing/light.h"
#include "acceleration/kdtreegpu.h"

#define NUM_REFLECTIONS 10
#define WARP_SIZE 32
#define N_THREAD 32

#define checkCudaErrors(val) check_cuda( (val), #val, __FILE__, __LINE__ )
void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        std::cerr << "CUDA error = " << static_cast<unsigned int>(result) << " at " <<
        file << ":" << line << " '" << func << "' \n";
        // Make sure we call CUDA Device Reset before exiting
        hipDeviceReset();
        exit(99);
    }
}

__device__ vec3 color(const ray& r, Camera c, StlObject obj, Lighting l) {
    // PHONG
    ray_hit rec;
    if (obj.hitTreeGPU(r, rec)) {
        vec3 illumination(0,0,0);
        // Light light;
        vec3 dirToCam = c.position - rec.p;
        for (int i = 0; i < l.count; i++) {
            Light light = l.lights[i];
            illumination += light.computePhong(rec.p, dirToCam, rec.normal, obj);
            illumination = clamp(illumination);
        }

        return illumination;
    }

    vec3 normalized = unit_vector(r.direction());
    float t = 0.5f*(normalized.x() + 1.0f);
    return (1.0f-t)*vec3(1.0, 1.0, 1.0) + t*vec3(0.5, 0.7, 1.0); 
}

__global__ void render(vec3 *frame, int x_max, int y_max, Camera camera, StlObject obj, Lighting l) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    int j = threadIdx.y + blockIdx.y * blockDim.y;

    if ((i >= x_max) || (j >= y_max)) {
        return;
    }

    int pixel_index = j * x_max + i;    
    float u = float(i) / float(x_max);
    float v = float(j) / float(y_max);

    ray toTrace = camera.make_ray(u, v);
    vec3 colorResult = color(toTrace, camera, obj, l);
    frame[pixel_index] = colorResult;
}

int main() {
    int n_cols = 6000;
    int n_rows = 4000;

    int tx = 8;
    int ty = 8;

    int num_pixels = n_cols * n_rows;
    size_t frame_size = num_pixels * sizeof(vec3);

    // allocating image frame
    vec3 *frame;
    hipMallocManaged((void **) &frame, frame_size);

    dim3 nthreads(tx, ty);
    dim3 nblocks(n_cols/tx + 1, n_rows/ty + 1);

    struct timeval startTime;
    struct timeval endTime;

    gettimeofday(&startTime, nullptr);
    std::vector<Triangle> triangles = StlParser::parseFile("examples/groot.stl");
    // std::vector<Triangle> triangles = StlParser::parseFile("examples/F-15.stl");
    // std::vector<Triangle> triangles = StlParser::parseFile("examples/pikachu.stl");
    gettimeofday(&endTime, nullptr);

    int millis = (endTime.tv_sec - startTime.tv_sec) * 1000 + (endTime.tv_usec - startTime.tv_usec) / 1000;

    std::cerr << "Parsing time: " << millis << "ms" << std::endl;

    size_t triangle_count = triangles.size();
    std::cerr << "Triangle count: " << triangle_count << std::endl;

    Triangle *object_h; //= triangles.data();
    // Triangle *object_d;
    
    checkCudaErrors(hipMallocManaged ( (void**) &object_h, sizeof(Triangle)*triangle_count));
    // checkCudaErrors(hipMemcpy (object_d, object_h, sizeof(Triangle)*triangle_count, hipMemcpyHostToDevice));  // TODO: Maybe use cuda host malloc? share the memory?
    std::memcpy(object_h, triangles.data(), sizeof(Triangle)*triangle_count);

    // TODO: think about what this looks like
    StlObject object(object_h, triangle_count);
    object.color = vec3(0.5,0.3,0);
    object.specular = vec3(1,1,1);
    object.shininess = 5;

    std::vector<Light> lightVec;
    // color, attenuation, dir for BMO
    // Light light1;
    // light1.makeDir(vec3(1, 0.6, 1), vec3(1,0,0), vec3(-1, 1, 0)); // PINK

    // Light light2;
    // light2.makeDir(vec3(0, 1, 0), vec3(1,0,0), vec3(-1, -1, 0)); // GREEN

    // Light light3;
    // light3.makeDir(vec3(0.75, 1, 1), vec3(1,0,0), vec3(0, 0, -1)); // CYAN

    // batman 
    // 45.2189 27.5099 134.38
    // -39.5991 -28.3567 0
    // Light light1;
    // light1.makeSpot(vec3(0.8, 0.9, 1), vec3(1,0,0), vec3(0, 0, 200), vec3(0,0,0), 0.8, 0.7); // light blue

    // Light light2;
    // light2.makeDir(vec3(1, 1, 0.8), vec3(1,0,0), vec3(-1, 1, 0)); // yellow

    // Light light3;
    // light3.makeDir(vec3(1, 0.4, 0.6), vec3(1,0,0), vec3(-1, -1, 0)); // red

    // groot
    // 45.2189 27.5099 134.38
    // -39.5991 -28.3567 0
    Light light1;
    light1.makeSpot(vec3(0.8, 0.9, 1), vec3(1,0,0), vec3(0, 0, 200), vec3(0,0,0), 0.5, 0.3); // light blue

    Light light2;
    light2.makeDir(vec3(0.1, 0.7, 0), vec3(1,0,0), vec3(-1, 1, 0)); // yellow

    Light light3;
    light3.makeDir(vec3(0.9, 0.85, 0), vec3(1,0,0), vec3(-1, -1, 0)); // red

    lightVec.push_back(light1);
    lightVec.push_back(light2);
    lightVec.push_back(light3);

    Light *lights;
    int lightCount = lightVec.size();
    checkCudaErrors(hipMallocManaged ( (void**) &lights, sizeof(Light) * lightCount));
    std::memcpy(lights, lightVec.data(), sizeof(Light) * lightCount);

    Lighting lighting(lights, lightCount);

    // copy over GPU Tree
    // copy over GPU TreeNodes
    // set pointers and fields
    TreeNodeGPU *treeNodesGPU_h;
    int node_count = object.treeGPU->node_count;
    checkCudaErrors(hipMallocManaged ( (void**) &treeNodesGPU_h, sizeof(TreeNodeGPU) * node_count));
    // checkCudaErrors(hipMemcpy (treeNodesGPU_d, object.treeGPU->nodes, sizeof(TreeNodeGPU)*node_count, hipMemcpyHostToDevice));  // TODO: Maybe use cuda host malloc? share the memory?
    std::memcpy(treeNodesGPU_h, object.treeGPU->nodes, sizeof(TreeNodeGPU)*node_count);

    KdTreeGPU treeGPU_h(object_h, triangle_count, treeNodesGPU_h, node_count);

    KdTreeGPU *treeGPU_u;
    checkCudaErrors(hipMallocManaged ( (void**) &treeGPU_u, sizeof(KdTreeGPU)));
    // checkCudaErrors(hipMemcpy (treeGPU_d, &treeGPU_h, sizeof(KdTreeGPU), hipMemcpyHostToDevice));
    std::memcpy(treeGPU_u,&treeGPU_h, sizeof(KdTreeGPU));

    // treeGPU_d->nodes = treeNodesGPU_d;
    // treeGPU_d->allTriangles = object_d;

    object.treeGPU = treeGPU_u;
    // object.triangles = object_d;

    // making camera

    vec3 bboxMin = object.tree->root->box.min;
    vec3 bboxMax = object.tree->root->box.max;
    std::cerr << "bboxMin:" << bboxMin << std::endl;
    std::cerr << "bboxMax:" << bboxMax << std::endl;
    vec3 centroid = (bboxMin + bboxMax) / 2.0f;
    std::cerr << "centroid:" << centroid << std::endl;

    // vec3 bmoPos(300, -300, 200);
    vec3 grootPos(300, -400, 150);

    // bmoPos *= 0.75;
    // Camera camera(vec3(bboxMax.x()*1.5f, 0, 0), centroid, (bboxMax.y() - bboxMin.y())*1.5f, (bboxMax.x()  - bboxMin.x())*1.5f);
    Camera camera(grootPos, centroid, 110, 165);

    std::cerr << "starting render" << std::endl;
    gettimeofday(&startTime, nullptr); 
    render<<<nblocks, nthreads>>>(frame, n_cols, n_rows, camera, object, lighting);
    hipError_t kernelError = hipGetLastError();
    if (kernelError != hipSuccess) {
        // Handle kernel launch error
        std::cerr << "Kernel launch error: " << hipGetErrorString(kernelError) << std::endl;
    }
    hipDeviceSynchronize();
    fflush(stdout);
    kernelError = hipGetLastError();
    if (kernelError != hipSuccess) {
        std::cerr << "Synchronize error: " << hipGetErrorString(kernelError) << std::endl;
    }
    gettimeofday(&endTime, nullptr);

    millis = (endTime.tv_sec - startTime.tv_sec) * 1000 + (endTime.tv_usec - startTime.tv_usec) / 1000;

    std::cerr << "Rendering time: " << millis << "ms" << std::endl;

    gettimeofday(&startTime, nullptr); 
    std::cout << "P3\n" << n_cols << " " << n_rows << "\n255\n";
    for (int j = n_rows-1; j >= 0; j--) {
        for (int i = 0; i < n_cols; i++) {
            size_t pixel_index = j*n_cols + i;
            int ir = int(255.99*frame[pixel_index].r());
            int ig = int(255.99*frame[pixel_index].g());
            int ib = int(255.99*frame[pixel_index].b());
            std::cout << ir << " " << ig << " " << ib << "\n";
        }
    }
    gettimeofday(&endTime, nullptr);

    millis = (endTime.tv_sec - startTime.tv_sec) * 1000 + (endTime.tv_usec - startTime.tv_usec) / 1000;

    std::cerr << "File Output Time: " << millis << "ms" << std::endl;

    hipFree(frame);
    hipFree(treeNodesGPU_h);
    hipFree(treeGPU_u);
    // hipFree(object_d);
}